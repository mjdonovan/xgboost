/*!
 * Copyright 2018-2022 XGBoost contributors
 */
#include "common.h"

namespace xgboost {
namespace common {

void SetDevice(std::int32_t device) {
  if (device >= 0) {
    dh::safe_cuda(hipSetDevice(device));
  }
}

int AllVisibleGPUs() {
  int n_visgpus = 0;
  try {
    // When compiled with CUDA but running on CPU only device,
    // hipGetDeviceCount will fail.
    std::cout <<"abs0" << std::endl;
    dh::safe_cuda(hipGetDeviceCount(&n_visgpus));
    std::cout <<"abs1" << std::endl;
  } catch (const dmlc::Error &e) {
    std::cout <<"abs2" << std::endl;
    hipGetLastError();  // reset error.
    std::cout << e.what() << std::endl;
    return 0;
  }
  std::cout <<"abs3" << std::endl;
  return n_visgpus;
}

}  // namespace common
}  // namespace xgboost
